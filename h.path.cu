#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <array>
#include <deque>
#include <iterator>
#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <thread>
#include <future>
#include <chrono>
#include <queue>

using std::cout;
using std::endl;
    
struct Entry {
  int weight;
  std::pair<int, int> coords;
};

struct Node {
  int val;
  Node* next1 = nullptr;
  Node* next2 = nullptr;
};

std::vector<Entry> owal;

constexpr int chessNo = 11;

void printOwal(const std::vector<Entry>& owal) {
  for (auto&& e : owal) {
      cout << "w:" << e.weight << " at (" << e.coords.first << "," << e.coords.second << ")" << endl;
  }
}

void printCom(const std::vector<int>& a) {
  for (auto&& e: a) {
      cout << e << " ";
  }
  cout << endl;
}

void printCount(const std::array<int, chessNo>& a) {
  cout << "Count: ";
  for (auto&& e : a)
      cout << e << " ";
  cout << endl;
}

inline std::array<Node, chessNo> genCycle(const std::vector<int>& com) {
  std::array<Node, chessNo> nodes;
  for (int i = 1; i <= chessNo; i++) {
      nodes[i - 1].val = i;
  }

  for (int i = 0; i < owal.size(); i++) {
      if (com[i]) {
          if (nodes[owal[i].coords.first - 1].next1 == nullptr) {
              nodes[owal[i].coords.first - 1].next1 = &nodes[owal[i].coords.second - 1];
          } else {
              nodes[owal[i].coords.first - 1].next2 = &nodes[owal[i].coords.second - 1];
          }
          if (nodes[owal[i].coords.second - 1].next1 == nullptr) {
              nodes[owal[i].coords.second - 1].next1 = &nodes[owal[i].coords.first - 1];
          } else {
              nodes[owal[i].coords.second - 1].next2 = &nodes[owal[i].coords.first - 1];
          }
      }
  }
  return nodes;
}

inline std::array<int, chessNo> genPath(const std::array<Node, chessNo>& nodes) {
  std::array<int, chessNo> path;
  const Node* n = &nodes[chessNo - 1];
  path[0] = n->val;
  int old1 = n->val;
  n = n->next1;
  path[1] = n->val;
  int old2 = n->val;
  for (int i = 2; i <= chessNo; i++) {
      old2 = n->val;
      if (old1 == n->next1->val)
          n = n->next2;
      else
          n = n->next1;
      path[i] = n->val;
      old1 = old2;
  }

  return path;
};

inline bool checkCycle(const std::array<int, chessNo>& cycle) {
  bool valid_first = true;
  bool valid_second = true;

  // check if continuous cycle
  if (std::find(cycle.begin()+1, cycle.end(), chessNo) != cycle.end()) {
      return false;
  }

  
  // ONE DIRECTION
  const int half = (chessNo - 1) / 2;
  for (int i = 1, n = 0; i < chessNo; i++) {
      if (cycle[(i) % chessNo] <= half)
          n++;
      else n--;
      if (n < 0) {
          valid_first = false;
          break;
      }
  }

  if (!valid_first) {
      for (int i = 1, n = 0; i < chessNo; i++) {
          if (cycle[(chessNo - i) % chessNo] <= half)
              n++;
          else n--;
          if (n < 0) {
              valid_second = false;
              break;
          }
      }
  }
  return valid_first || valid_second;
}

void printNodes(const std::array<Node, chessNo> nodes) {
  cout << "Cycle: ";
  const Node* n = &nodes[chessNo - 1];
  cout << n->val << " ";
  int old1 = n->val;
  n = n->next1;
  cout << n->val << " ";
  int old2 = n->val;
  for (int i = 0; i < chessNo - 2; i++) {
      old2 = n->val;
      if (old1 == n->next1->val)
          n = n->next2;
      else
          n = n->next1;
      cout << n->val << " ";
      old1 = old2;
  }
  cout << endl;
}

void printPath(const std::array<int, chessNo> p) {
  cout << "Cycle: ";
  for (auto&& e : p) {
      cout << e << " ";
  }
  cout << endl;
}

__global__ inline bool checkCombination(const std::vector<int>& com) {
  std::array<int, chessNo> count{};
  for (int i = 0; i < owal.size(); i++) {
      if (com[i]) {
          count[owal[i].coords.first - 1]++;
          count[owal[i].coords.second - 1]++;
      }
  }
//   printCount(count);
  for (int i = 0; i < chessNo; i++) {
      if (count[i] != 2) break;
      else if (i == chessNo - 1) {
          std::array<Node, chessNo> nodes = genCycle(com);
          // printCom(com);
          // printCount(count);
          std::array<int, chessNo> path = genPath(nodes);
          // printPath(path);
          if (checkCycle(path)) {
              cout << "FOUND." << endl;
              printPath(path);
              printCom(com);
              int w = 0;
              for (int j = 0; j < owal.size(); j++) {
                  if (com[j]) {
                      w += owal[j].weight;
                  }
              }
              cout << "Total weight: " << w << endl;
              return true;
          }

      }
  }
  return false;
}

#define BLOCK_DIM   128
#define GRID_DIM    32

__global__ void Find(std::vector<int>* coms, int* result)
{
  int bx = blockIdx.x;

  // Thread index
  int tx = threadIdx.x;

  std::vector<int>& com = coms[bx*BLOCK_DIM+tx];
  bool _result = checkCombination(com);
  __shared__ uint32_t found_id;
  found_id = BLOCK_DIM*GRID_DIM+1; 
  __syncthreads();
  if(_result){
      found_id = min(found_id,bx*BLOCK_DIM+tx);
  }
  __syncthreads();
  *result = found_id;
}

int main()
{
	
  std::srand(std::time(nullptr));

    /** Setup OWAL */
  for (int i = 1, temp = 0; i <= chessNo; i++) {
    for (int j = i + 1; j <= chessNo; j++) {
      cout << "(" << i << "," << j << ")";
      // cin >> temp; 
      temp = 1+std::rand()%50;
      owal.push_back({temp, {i, j}});
    }
  }
  sort(owal.begin(), owal.end(), [](const Entry& a, const Entry& b) {
    return a.weight < b.weight;
  });
  printOwal(owal);

  /** Setup k-combination */
  std::vector<int> com = {};

  for (int i = 0; i < chessNo; i++)
    com.push_back(1);
  for (int i = 0; i < owal.size() - chessNo; i++)
    com.push_back(0);

  std::next_permutation(com.begin(), com.end());
  std::vector<int> original_com = com;


  /** for each k-combination, O(n) per cycle */
  int cnt = 0;
  
  /** for CUDA*/
  constexpr int len = GRID_DIM*BLOCK_DIM;   //length of coms to handle in parallel
  std::vector<int>* d_coms = nullptr;  //coms array store in gpu(device)
  std::vector<int> h_coms[len];  //coms array store in host
  int* d_found_id;
  int h_found_id;
  hipMalloc((void**) &d_coms, sizeof(com) * len);
  hipMalloc((void**) &d_found_id, sizeof(int));

  dim3 dimBlock(BLOCK_DIM);
	dim3 dimGrid(GRID_DIM);

  int push_len = 0;

  do {
    if(push_len<len){
      prev_permutation(com.begin(), com.end());
      h_coms[push_len]=com;
      push_len++;
    } else {
      hipMemcpy(d_coms, h_coms, len* sizeof(com), hipMemcpyHostToDevice);
      Find<<<dimGrid,dimBlock>>>(d_coms,d_found_id);
      hipMemcpy(&h_found_id, d_found_id, sizeof(int),hipMemcpyDeviceToHost);
      if(h_found_id!=-1){
        std::vector<int>& temp = h_coms[h_found_id];cout<<endl;
        for(int i:temp)cout<<i<<" ";
        cout<<endl;
        break;
      }
      push_len = 0;
    }
    if (++cnt % 10000000 == 0) cout << cnt << endl;
    // if (checkCombination(com)) break;

  } while (com != original_com);

	return 0;
}

